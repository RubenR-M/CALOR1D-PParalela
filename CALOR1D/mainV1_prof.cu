#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include "gnuplot.h"
#include <fstream>
#include <sstream>
#include "nvToolsExt.h"
#include "math.h"
using namespace std;

int Nt = 93;
int Nx = 10;
float dt = 0.01;
float dx = 0.1;
float T_0[] = {20, 30, 0, 0, 0, 0, 0, 0, 0, 30, 20};
float T_izq = 50, T_der = 50;
float k = 0.16;
/*
 * nvcc mainV1_prof.cu -o mainV1_prof -arch=sm_50 -lnvToolsExt
 * nsys profile -o mainV1 mainV1_prof
 */
__global__ void temperatura(float *d_T, float *d_t0, int Nx, int Nt, float T_der, float T_izq, float k)
{
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    if (idx == Nx - 1 && idy > 0)
    {
        d_T[threadIdx.y * Nx + threadIdx.x] = T_izq;
    }
    if (idx == 0 && idy > 0)
    {
        d_T[threadIdx.y * Nx + threadIdx.x] = T_der;
    }
    if (idy == 0)
    {
        d_T[threadIdx.y * Nx + threadIdx.x] = d_t0[threadIdx.x];
    }
    __syncthreads();
    if (idy > 0 && (idx > 0 && idx < Nx - 1))
    {
        // T[f * Nx + c] = T[(f - 1) * Nx + c] + k * (T[(f - 1) * Nx + c + 1] - 2 * T[(f - 1) * Nx + c] + T[(f - 1) * Nx + c - 1]);

        for (int i = 1; i < Nt; i++)
        {

            float a = (float)(d_T[(i - 1) * Nx + threadIdx.x]);
            float b = (float)(d_T[(i - 1) * Nx + threadIdx.x + 1]);
            float c = (float)(d_T[(i - 1) * Nx + threadIdx.x]);
            float d = (float)(d_T[(i - 1) * Nx + threadIdx.x - 1]);

            d_T[i * Nx + threadIdx.x] = (float)(a + k * (b - 2 * c + d));
        }
    }
}

void read(string path)
{
    vector<double> T_0;
    string strT_0;

    path = path;
    ifstream fin;
    fin.open(path);
    if (fin.is_open())
    {
        fin >> Nt >> Nx >> dt >> dx >> strT_0 >> T_izq >> T_der >> k;
        fin.close();
    }

    while (1) // Use a while loop, "i" isn't doing anything for you
    {
        if (strT_0.find(',') != std::string::npos) // if comman not found find return string::npos
        {
            double value;
            istringstream(strT_0) >> value;
            T_0.push_back(value);
            strT_0.erase(0, strT_0.find(',') + 1); // Erase all element including comma
        }
        else
            break; // Come out of loop
    }
}

void graph(float *h_T, vector<float> X, int Nx, int Nt)
{
    string nombreArchivo = "datos.dat";
    ofstream archivo;

    cout << setprecision(3) << fixed;
    cout << "t/x      | ";
    for (int i = 0; i < Nx; i++) // creacion del vector de longitudes espaciado
    {
        if (i == 0 || i == Nx - 1)
        {
            cout << to_string(X[i]) << "000  ";
        }
        else
        {
            cout << to_string(X[i]) << "00  ";
        }
    }
    cout << endl;
    cout << "------------------------------------------------------------------" << endl;

    vector<float> tiempo;
    for (int i = 0; i < Nx * Nt; i++)
    {
        if (i == 0)
        {
            cout << to_string(i * dt) << " | ";
        }
        if (i == 0 || i == Nx - 1)
        {
            cout << to_string(h_T[i]) << "00  ";
        }
        else if (to_string(h_T[i]).size() < 12)
        {
            cout << to_string(h_T[i]) << "0  ";
        }
        else
        {
            cout << to_string(h_T[i]) << "  ";
        }

        if (i != Nx * Nt - 1 && (i + 1) % Nx == 0)
        {
            if (tiempo.size() == 0)
            {
                tiempo.push_back(0);
            }
            cout << endl;
            cout << to_string(((i + 1) * dt) / Nx) << " | ";
            tiempo.push_back(((i + 1) * dt) / Nx);
        }
    }
    cout << endl;

    for (int i = 0; i < (Nt)-1; i++)
    {
        for (int j = 0; j < Nx; j++)
        {
            X.push_back(j * dx);
        }
    }
    vector<float> t;
    for (int j = 0; j < Nt; j++)
    {
        for (int i = 0; i < Nx; i++)
        {
            t.push_back(tiempo[j]);
        }
    }
    archivo.open(nombreArchivo.c_str(), fstream::out);
    for (int i = 0; i < X.size(); i++)
    {
        archivo << X[i] << " " << t[i] << " " << h_T[i] << endl;
    }
    archivo.close();

    gnuplot p;
    p("set view map");
    p("set dgrid3d");
    p("set pm3d interpolate 0,0");
    p("splot 'datos.dat' using 1:2:3 with pm3d");
}

int main(int argc, char **argv)
{
    hipEvent_t start, stop;
    if (argc > 1)
    {
        string path = (string)argv[1];
        read(path);
    }
    else
    {
        cout << "No entry found!!\nDefault Values: " << endl;
        cout << "Nt: " << Nt << endl
             << "Nx: " << Nx << endl
             << "dt: " << dt << endl
             << "dx: " << dx << endl
             << "T_der: " << T_der << endl
             << "T_izq: " << T_izq << endl
             << "k: " << k << endl;
        cout << "Tempaturas iniciales" << endl;
        cout << "[ ";
        for (int i = 0; i < Nx + 1; i++)
        {
            printf("%f ", T_0[i]);
        }
        cout << "]" << endl;
    }
    Nx = Nx + 1;
    // vector de la vara lleno de 0's
    float *h_T;
    // vector de las temperaturas iniciales
    float *h_t0;
    float millis;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    nvtxRangePush("generate data cpu");
    hipEventRecord(start);
    hipHostMalloc((void **)&h_T, sizeof(float) * Nx * Nt);
    hipHostMalloc((void **)&h_t0, sizeof(float) * Nx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);

    vector<float> X;
    for (int j = 0; j < Nx; ++j)
    {
        X.push_back(j * dx); // PARA GRAFICAR
        h_t0[j] = T_0[j];
    }

    for (int i = 0; i < Nt; ++i)
    {
        for (int j = 0; j < Nx; ++j)
        {
            h_T[i * Nx + j] = 0.0;
        }
    }
    // vector de la vara lleno de 0's en Grafica
    float *d_T;
    // vector de las temperaturas en Grafica
    float *d_t0;

    nvtxRangePush("generate data gpu");
    hipEventRecord(start);
    hipMalloc((void **)&d_T, sizeof(float) * Nx * Nt);
    hipMalloc((void **)&d_t0, sizeof(float) * Nx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);

    // hipEventRecord(start);
    nvtxRangePush("Transfer to GPU");
    hipEventRecord(start);
    hipMemcpy(d_T, h_T, sizeof(float) * Nx * Nt, hipMemcpyHostToDevice);
    hipMemcpy(d_t0, h_t0, sizeof(float) * Nx, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);

    // hipEventElapsedTime(&millis, start, stop);
    // unsigned int grid_rows = Nt;
    // unsigned int grid_cols = Nx;

    dim3 dimGrid(Nx, Nt);
    nvtxRangePush("Calculating Temperature");
    hipEventRecord(start);
    temperatura<<<1, dimGrid>>>(d_T, d_t0, Nx, Nt, T_der, T_izq, k);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);
    // hipDeviceSynchronize(); //No necesario por el EventSync
    

    

    
    nvtxRangePush("GPU to CPU");
    hipEventRecord(start);
    hipMemcpy(h_T, d_T, sizeof(float) * Nx * Nt, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);

    graph(h_T, X, Nx, Nt);
    nvtxRangePush("free memory");
    hipEventRecord(start);
    hipFree(d_T);
    hipFree(d_t0);
    hipHostFree(h_T);
    hipHostFree(h_t0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    nvtxRangePop();
    hipEventElapsedTime(&millis, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
